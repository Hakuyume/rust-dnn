
#include <hip/hip_runtime.h>
#define FOREACH(index, length) for (size_t index = threadIdx.x + blockIdx.x * blockDim.x; index < length; index += gridDim.x * blockDim.x)

extern "C" __global__ void relu_forward_inplace_f(float *src, size_t len)
{
  FOREACH(i, len) {
    if (src[i] < 0)
      src[i] = 0;
  }
}
